
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 2
int get_max_size (int a, int d)
{
    int temp = a/d;
    if (a%d != 0)
    {
        temp = temp+1;
    }
    return temp;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void fixup(unsigned int *input, unsigned int *aux, int len) {
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
    if (blockIdx.x > 0) {
       if (start + t < len)
          input[start + t] += aux[blockIdx.x ];
       if (start + BLOCK_SIZE + t < len)
          input[start + BLOCK_SIZE + t] += aux[blockIdx.x ];
    }
}

__global__ void scanPart1 (unsigned int* input,
			   unsigned int* output,
			   unsigned int* aux,
			   int numElems)
{
	extern __shared__ unsigned int sdata[];
	//int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	int myLocalId = threadIdx.x;
	int start = 2 * blockIdx.x * BLOCK_SIZE; //Each block reads 2*BLOCK_SIZE so idx*this value is total inputs read
	int lastReadValue = 0;
	//Input Read
	if (start + myLocalId < numElems)
	{
       		sdata[myLocalId] = input[start + myLocalId];
	}
	else
	{
       		sdata[myLocalId] = 0;
	}
	if (start + BLOCK_SIZE + myLocalId  < numElems)
	{
       		sdata[BLOCK_SIZE + myLocalId] = input[start + BLOCK_SIZE + myLocalId];
	}
    	else
	{
       		sdata[BLOCK_SIZE + myLocalId] = 0;
	}
	__syncthreads();
	lastReadValue = sdata[2*BLOCK_SIZE-1];
	
	//Reduction
	int stride;
	for (stride = 1; stride <= BLOCK_SIZE; stride <<= 1) 
	{
       		int index = (myLocalId + 1) * stride * 2 - 1;
	        if (index < 2 * BLOCK_SIZE)
 	        	sdata[index] += sdata[index - stride];
	        __syncthreads();
	}
	if (myLocalId == 0)
	{
		sdata[2*BLOCK_SIZE-1] = 0;
	}
	__syncthreads();
	// Post reduction
	for (stride = BLOCK_SIZE ; stride; stride >>= 1) 
	{
       		int index = (myLocalId + 1) * stride * 2 - 1;
	        //if (index + stride < 2 * BLOCK_SIZE)
	        if (index < 2 * BLOCK_SIZE)
		{
//			unsigned int temp = sdata[index+stride];
//         		sdata[index + stride] += sdata[index];
//			sdata[index] = temp;
			unsigned int temp = sdata[index];
         		sdata[index] += sdata[index-stride];
			sdata[index-stride] = temp;
		}
	        __syncthreads();
    	}

	if (start + myLocalId < numElems)
	       	output[start + myLocalId] = sdata[myLocalId];
	if (start + BLOCK_SIZE + myLocalId < numElems)
        	output[start + BLOCK_SIZE + myLocalId] = sdata[BLOCK_SIZE + myLocalId];

	if (myLocalId == 0 && aux!=NULL)
       		aux[blockIdx.x] = sdata[2 * BLOCK_SIZE - 1] + lastReadValue;
}

__global__ void splitInput(int compareAndValue,
			   unsigned int* input,
			   unsigned int* output,
			   int maxElements)
{
	int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	if (myGlobalId >= maxElements)
	{
		return;
	}
	if(((input[myGlobalId] & compareAndValue)) > 0)
	{
		printf("%d. %d & %d is 0\n",myGlobalId,input[myGlobalId],compareAndValue);
		output[myGlobalId] = 0;
	}
	else
	{
		printf("%d. %d & %d is 1\n",myGlobalId,input[myGlobalId],compareAndValue);
		output[myGlobalId] = 1;
	}
	printf("%d. %d\n",myGlobalId,input[myGlobalId]);
}

__global__ void possibleLocations (unsigned int* input,
				   unsigned int* input_scan,
				   unsigned int* output,
				   unsigned int numElems,
				   unsigned int compareAndValue)
{
	int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	int myLocalId = threadIdx.x;
	int start = 2 * blockIdx.x * BLOCK_SIZE; //Each block reads 2*BLOCK_SIZE so idx*this value is total inputs read
	int total = input_scan[numElems-1] + (((input[numElems-1] & compareAndValue) > 0)?0:1);
	printf("Total %d\n",total);
	if (myLocalId + start < numElems)
	{
		//output[myGlobalId] = myGlobalId - input_scan[myGlobalId] + total;
		output[start + myLocalId] = start + myLocalId  - input_scan[start + myLocalId] + total;
		printf("%d. %d might go to %d\n",start + myLocalId,input[myLocalId + start], output[start + myLocalId]);
	}
	if (myLocalId + start + BLOCK_SIZE < numElems)
	{
		output[start + myLocalId + BLOCK_SIZE] = start + myLocalId + BLOCK_SIZE - input_scan[start + myLocalId + BLOCK_SIZE] + total ;
		printf("%d. %d might go to %d\n",start + myLocalId,input[myLocalId + start+BLOCK_SIZE], output[start + myLocalId + BLOCK_SIZE]);
	}
	
}


__global__ void finalLocations (   unsigned int* input,
				   unsigned int* input_scan,
				   unsigned int* input_vals,
				   unsigned int* d_setOneIfOne,
				   unsigned int* output,
				   unsigned int numElems)
{
	int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	int myLocalId = threadIdx.x;
	int start = 2 * blockIdx.x * BLOCK_SIZE; //Each block reads 2*BLOCK_SIZE so idx*this value is total inputs read
	if (myLocalId + start < numElems)
	{
		if (d_setOneIfOne[myLocalId + start] == 0)
		{
			output[input[myLocalId + start]] = input_vals[myLocalId + start];
			printf("%d. %d goes to %d\n",myGlobalId, input_vals[myLocalId + start], input[myLocalId + start]);
		}	
		else
		{
			output[input_scan[myLocalId + start]] = input_vals[myLocalId + start];
			printf("%d. %d goes to %d\n",myGlobalId, input_vals[myLocalId + start], input_scan[myLocalId + start]);
		}
	}
	if (myLocalId + start + BLOCK_SIZE < numElems)
	{	
		if (d_setOneIfOne[myLocalId + start + BLOCK_SIZE] == 0)
		{
			output[input[myLocalId + start + BLOCK_SIZE]] = input_vals[myLocalId + start + BLOCK_SIZE];
			printf("%d. %d goes to %d\n",myGlobalId, input_vals[myLocalId + start+ BLOCK_SIZE], input[myLocalId + start +BLOCK_SIZE]);
		}	
		else
		{
			output[input_scan[myLocalId + start + BLOCK_SIZE]] = input_vals[myLocalId + start + BLOCK_SIZE];
			printf("%d. %d goes to %d\n", myGlobalId, input_vals[myLocalId + start+BLOCK_SIZE] , input_scan[myLocalId + start +BLOCK_SIZE]);
		}
	}
}

int main()
{
	unsigned int h_inputVals[10] = {3, 4, 1, 2, 7, 6, 5, 0, 9, 8};
  	unsigned int numElems = 10;
        unsigned int  h_bins[2];
	int histo_size = sizeof(unsigned int)*2;
	unsigned int* d_inputVals;
	gpuErrchk(hipMalloc(&d_inputVals, numElems*sizeof(numElems)));
	gpuErrchk(hipMemcpy(d_inputVals, h_inputVals, numElems*sizeof(numElems), hipMemcpyHostToDevice));
        unsigned int* d_bins;
        gpuErrchk(hipMalloc(&d_bins, histo_size));
     	unsigned int* d_setOneIfOne;
     	unsigned int* d_possibleLocations;
     	unsigned int* d_finalLocations;
     	unsigned int* d_scan;
     	unsigned int* h_scan;
	h_scan = (unsigned int*)malloc(numElems*sizeof(numElems));
	gpuErrchk(hipMalloc(&d_setOneIfOne, numElems*sizeof(numElems)));
	gpuErrchk(hipMalloc(&d_scan, numElems*sizeof(numElems)));
	gpuErrchk(hipMalloc(&d_possibleLocations, numElems*sizeof(numElems)));
	for (int i=0;i<10;i++)
	{
	       printf("%d ", h_inputVals[i]);
	}
	printf("\n");
 
     	unsigned int* h_setOneIfOne;
	h_setOneIfOne = (unsigned int*)malloc(numElems*sizeof(numElems));
        for (int i=0;i<4;i++)
        {
		 gpuErrchk(hipMalloc(&d_finalLocations, numElems*sizeof(numElems)));
		 printf("Round %d\n",i);
 	       	 gpuErrchk(hipMemset(d_bins, 0, histo_size));
 	       	 gpuErrchk(hipMemset(d_setOneIfOne,0, numElems*sizeof(numElems)));
 	       	 gpuErrchk(hipMemset(d_scan,0, numElems*sizeof(numElems)));
	         int compareAndValue = 1 << i;
		 int numberThreadPerBlock = 512;
		 dim3 blockDim_si(numberThreadPerBlock);
		 dim3 gridDim_si(get_max_size(numElems,numberThreadPerBlock));
		 splitInput<<<gridDim_si,blockDim_si>>>(compareAndValue, d_inputVals, d_setOneIfOne, numElems);
	         gpuErrchk(hipMemcpy(h_setOneIfOne, d_setOneIfOne, numElems*sizeof(numElems), hipMemcpyDeviceToHost));
		 for (int i=0;i<10;i++)
		 {
			printf("%d ", h_setOneIfOne[i]);
			h_setOneIfOne[i] = 0;
		 }
		 printf("\n");
		 dim3 blockDim_sp(BLOCK_SIZE);
		 dim3 gridDim_sp(get_max_size(numElems,2*BLOCK_SIZE));
     		 unsigned int* d_aux;
     		 unsigned int* d_aux_scan;
		 unsigned int* h_aux;
	 	 gpuErrchk(hipMalloc(&d_aux, get_max_size(numElems,2*BLOCK_SIZE)*sizeof(unsigned int)));
	 	 gpuErrchk(hipMalloc(&d_aux_scan, get_max_size(numElems,2*BLOCK_SIZE)*sizeof(unsigned int)));
		 h_aux = (unsigned int*)malloc(get_max_size(numElems,2*BLOCK_SIZE)*sizeof(unsigned int));
		 
	//         gpuErrchk(cudaMemcpy(d_scan, d_setOneIfOne, numElems*sizeof(numElems), cudaMemcpyDeviceToDevice));
		 printf ("Size of Kernel is Grid - %d, Block - %d\n",gridDim_sp.x,blockDim_sp.x);
		 scanPart1<<<gridDim_sp,blockDim_sp,BLOCK_SIZE*2*sizeof(unsigned int)>>> (d_setOneIfOne,d_scan,d_aux,numElems);	
	         gpuErrchk(hipMemcpy(h_scan, d_scan, numElems*sizeof(numElems), hipMemcpyDeviceToHost));	
		 for (int i=0;i<10;i++)
		 {
			printf("%d ", h_scan[i]);
			h_scan[i] = 0;
		 }
		 printf("\n");
		 dim3 blockDim_sp2(get_max_size(numElems,2*BLOCK_SIZE));
	         gpuErrchk(hipMemcpy(h_aux, d_aux, blockDim_sp2.x*sizeof(unsigned int), hipMemcpyDeviceToHost));	
		 for (int i=0;i<blockDim_sp2.x;i++)
		 {
			printf("%d ", h_aux[i]);
			h_aux[i] = 0;
		 }
		 printf("\n");

		 printf ("Size of Kernel is Grid - 1, Block - %d\n",blockDim_sp2.x);
		 scanPart1<<<1,blockDim_sp2,BLOCK_SIZE*2*sizeof(unsigned int)>>>(d_aux,d_aux_scan,NULL,blockDim_sp2.x);	
	         gpuErrchk(hipMemcpy(h_aux, d_aux_scan, blockDim_sp2.x*sizeof(unsigned int), hipMemcpyDeviceToHost));	
		 for (int i=0;i<blockDim_sp2.x;i++)
		 {
			printf("%d ", h_aux[i]);
			h_aux[i] = 0;
		 }
		 printf("\n");
		 printf ("Size of Kernel is Grid - %d, Block - %d\n",gridDim_sp.x,blockDim_sp.x);
		 fixup<<<gridDim_sp,blockDim_sp>>>(d_scan,d_aux_scan,numElems);
	         gpuErrchk(hipMemcpy(h_scan, d_scan, numElems*sizeof(numElems), hipMemcpyDeviceToHost));	
		 for (int i=0;i<10;i++)
		 {
			printf("%d ", h_scan[i]);
			h_scan[i] = 0;
		 }
		 printf("\n");
		 printf ("Size of Kernel is Grid - %d, Block - %d\n",gridDim_sp.x,blockDim_sp.x);
		 possibleLocations<<<gridDim_sp,blockDim_sp>>>(d_inputVals,d_scan, d_possibleLocations, numElems, compareAndValue);
                 gpuErrchk(hipMemcpy(h_setOneIfOne, d_possibleLocations, numElems*sizeof(numElems), hipMemcpyDeviceToHost));
		 printf ("Possible Locations are \n");
		 for (int i=0;i<10;i++)
		 {
			printf("%d ", h_setOneIfOne[i]);
			h_setOneIfOne[i] = 0;
		 }
		 printf ("\n");
		 finalLocations<<<gridDim_sp,blockDim_sp>>>(d_possibleLocations,d_scan,d_inputVals, d_setOneIfOne, d_finalLocations,numElems);
	 	 hipDeviceSynchronize(); 
		 gpuErrchk(hipFree(d_inputVals));
		 d_inputVals = d_finalLocations; 
                 gpuErrchk(hipMemcpy(h_setOneIfOne, d_finalLocations, numElems*sizeof(numElems), hipMemcpyDeviceToHost));
		 printf ("\nFinal Positions are \n");
		 for (int i=0;i<10;i++)
		 {
		        printf("%d ", h_setOneIfOne[i]);
		        h_setOneIfOne[i] = 0;
		 }

		 printf("\n******************************************\n");
	         //printf("Histogram Values - %d %d %d %d %d \n", h_bins[0], h_bins[1], h_bins[0]+h_bins[1], numElems, compareAndValue);
        }
        gpuErrchk(hipFree(d_bins));
	gpuErrchk(hipFree(d_setOneIfOne));
	free(h_setOneIfOne);
	gpuErrchk(hipFree(d_possibleLocations));
	free(h_scan);
	return 0;
}

