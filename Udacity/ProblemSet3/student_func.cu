#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"
#include <limits.h>
#include <float.h>
#include <math.h>
#include <stdio.h>

int get_max_size(int n, int d) {
    int size = n/d;
    if (n%d !=0 )
    {
        size = size+1;
    }
    return size;
}

__global__ 
void scan_kernel(unsigned int* d_bins, int size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if(mid >= size)
        return;
    
    for(int s = 1; s <= size; s *= 2) {
          int spot = mid - s; 
         
          unsigned int val = 0;
          if(spot >= 0)
              val = d_bins[spot];
          __syncthreads();
          if(spot >= 0)
              d_bins[mid] += val;
          __syncthreads();

    }
}

__global__ void histogramReduce (int numBins,
                                 unsigned int *input,
                                 unsigned int *output,
                                 int size
                                )
{
    extern __shared__ unsigned int sdata3[];
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    for (int i=0;i<numBins;i++)
    {
        int index = myX*numBins+i;
        if (myX >= size)
        {
            sdata3[tid*numBins+i] = 0;
        }
        else
        {
            sdata3[tid*numBins+i] = input[index];
        }
    }
    __syncthreads();
    
    if (myX >= size)
    {
        if (tid == 0)
        {
            for (int i=0;i < numBins;i++)
            {
                output[blockIdx.x*numBins+i] = 0;
            }
        }
        return;
    }
    
    for (unsigned int s = blockDim.x/2; s > 0; s/=2)
    {
        if(tid < s)
        {
            for (int i=0;i<numBins;i++)
            {
                sdata3[tid*numBins+i] = sdata3[tid*numBins+i] + sdata3[tid*numBins+i+s*numBins];
            }
        }
        __syncthreads();
    }
    if (tid==0)
    {
        for (int i=0;i<numBins;i++)
        {
            //printf("Writing %d for bin value %d\n",sdata3[i],i);
            output[blockIdx.x*numBins+i] = sdata3[i];
        }
    }
}

__global__ void localHistograms (const float *input,
                                 unsigned int *output,
                                 const int size,
                                 int numBins,
                                 float min_logLum,
                                 float range,
                                 int perThreadReads)
{
    extern __shared__ unsigned int sdata2[];
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    //int tid = threadIdx.x;
    //printf("myX is %d and myX*perThreadReads is %d and size is %d\n",myX,myX*perThreadReads,size);
    for (int i = 0 ; i < numBins; i++)
    {
        sdata2[i] = 0;
    }

    for (int i=0;i<perThreadReads;i++)
    {
        if (myX*perThreadReads+i < size)
        {
            double lum = input[myX*perThreadReads+i];
            int bin = (lum - min_logLum) / range * numBins;
            sdata2[bin] = sdata2[bin]+1;
        }
    }
    for (int i = 0 ; i < numBins; i++)
    {
        output[blockIdx.x*numBins+i] = sdata2[i];
    }
}
__global__ void minmaxLuminance(const float* input,
                             float* output,
                             const int size,
                             int minmax)
{
    //minmax = 0 - min
    //min max = 1 - max
    
    extern __shared__ float sdata[];
    
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    if (myX >= size)
    {
        if (minmax == 0)
        {
            sdata[tid] = FLT_MAX;
        }
        else
        {
            sdata[tid] = -FLT_MAX;
        }
    }
    else
    {
        sdata[tid] = input[myX];
    }
    __syncthreads();

    if (myX >= size)
    {
        if (tid == 0)
        {
            if (minmax == 0)
            {
                output[blockIdx.x] = FLT_MAX;
            }
            else
            {
                output[blockIdx.x] = -FLT_MAX;
            }
        }
        return;
    }

    for (unsigned int s = blockDim.x/2; s > 0; s/=2)
    {
        if (tid < s)
        {
            if (minmax == 0)
            {
                sdata[tid] = min(sdata[tid],sdata[tid + s]);
            }
            else
            {
                sdata[tid] = max(sdata[tid],sdata[tid + s]);
            }
        }
        __syncthreads();
    }
    //printf("BlockIdx.x - %d\n",blockIdx.x);
    if (tid == 0)
    {
        output[blockIdx.x] = sdata[0];
    }
}

void calculateBins(unsigned int *d_finalHist, 
              const float* const d_logLuminance,
              const size_t numBins,
              const size_t numRows,
              const size_t numCols,
              float min_logLum,
              float range)
{
    unsigned int *d_histBins;
    int numThreadsPerBlock = 512;
    int size = numRows*numCols;
    printf("Size is %d\n",size);
    dim3 blockDim(1);
    dim3 gridDim(numThreadsPerBlock);
    int sizeOfBins = numBins*sizeof(unsigned int)*numThreadsPerBlock;
    int perThreadReads = get_max_size(size,numThreadsPerBlock);
    checkCudaErrors(hipMalloc(&d_histBins, sizeOfBins));
    localHistograms<<<gridDim,blockDim,numBins*sizeof(unsigned int)>>>(d_logLuminance,d_histBins,size,numBins,min_logLum,range,perThreadReads);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    size = numThreadsPerBlock;
    int localHistThreadsPerBlock = 8;
    dim3 blockDimLocalHist(localHistThreadsPerBlock);
    
    unsigned int* d_curr_in;
    unsigned int* d_curr_out;
    checkCudaErrors(hipMalloc(&d_curr_in, numBins*sizeof(unsigned int)*numThreadsPerBlock));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_histBins, numBins*sizeof(unsigned int)*numThreadsPerBlock, hipMemcpyDeviceToDevice));
    
    //unsigned int h_temp[1024*512];
    //checkCudaErrors(hipMemcpy(&h_temp, d_histBins,  numBins*sizeof(unsigned int)*numThreadsPerBlock, hipMemcpyDeviceToHost));
    //for(int i = 1024*511; i < 1024*512; i++)
    //    printf("hist out %d\n", h_temp[i]);
    while (size != 1 )
    {
        dim3 gridDimLocalHist(get_max_size(size,localHistThreadsPerBlock));
        printf("Histogram Reduce - Block Size - %d with size - %d\n",gridDimLocalHist.x,size);
        //Allocate d_curr_out here
        checkCudaErrors(hipMalloc(&d_curr_out, numBins*sizeof(unsigned int) * get_max_size(size,localHistThreadsPerBlock)));
        
        //Call Kernel
        histogramReduce<<<gridDimLocalHist,blockDimLocalHist,numBins*sizeof(unsigned int)*localHistThreadsPerBlock>>>(numBins,d_curr_in,d_curr_out,size);
        
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        //Update size here
        size = get_max_size(size,localHistThreadsPerBlock);
    }
    checkCudaErrors(hipFree(d_histBins));
    checkCudaErrors(hipMemcpy(d_finalHist, d_curr_out, sizeof(unsigned int)*numBins, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipFree(d_curr_out));
}

float findRange (const float* const d_logLuminance, 
                 const size_t numRows,
                 const size_t numCols,
                 int minmax)
{
    float returnValue;
    //Write code here
    int size = numRows*numCols;
    
    int numThreadsInBlock = 32;
    dim3 threadDim(numThreadsInBlock);
    
    float *d_curr_out;
    float *d_curr_in;
    checkCudaErrors(hipMalloc(&d_curr_in, sizeof(float) * size));    
    checkCudaErrors(hipMemcpy(d_curr_in, d_logLuminance, sizeof(float) * size, hipMemcpyDeviceToDevice));
    
    int shmem_size = numThreadsInBlock;//Number of threads in a block;
    while (size != 1 )
    {
        dim3 blockDim(get_max_size(size,numThreadsInBlock));
        printf("MinMax - Block Size - %d\n",blockDim.x);
        //Allocate d_curr_out here
        checkCudaErrors(hipMalloc(&d_curr_out, sizeof(float) * get_max_size(size,numThreadsInBlock)));
        
        //Call Kernel
        minmaxLuminance<<<blockDim,threadDim,shmem_size>>>(d_curr_in,d_curr_out, size,minmax);
        
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        //Update size here
        size = get_max_size(size,numThreadsInBlock);
    }
    
    checkCudaErrors(hipMemcpy(&returnValue, d_curr_out, sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_curr_out));
    return returnValue;
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
    
    // 1
    min_logLum = findRange(d_logLuminance,numRows,numCols,0);
    max_logLum = findRange(d_logLuminance,numRows,numCols,1);
    printf("got min of %f\n", min_logLum);
    printf("got max of %f\n", max_logLum);
    // 2
    float range = max_logLum - min_logLum;
    printf("got range of %f\n", range);
    // 3
    unsigned int *d_histBins;
    int sizeOfBins = sizeof(unsigned int)*numBins;
    checkCudaErrors(hipMalloc(&d_histBins, sizeOfBins));
    calculateBins(d_histBins, d_logLuminance, numBins,numRows,numCols,min_logLum,range);
    unsigned int h_out[100];
    hipMemcpy(&h_out, d_histBins, sizeof(unsigned int)*100, hipMemcpyDeviceToHost);
    
    // 4
    
    dim3 thread_dim(1024);
    dim3 scan_block_dim(get_max_size(numBins, thread_dim.x));
    scan_kernel<<<scan_block_dim, thread_dim>>>(d_histBins, numBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());    
    hipMemcpy(d_cdf, d_histBins, sizeOfBins, hipMemcpyDeviceToDevice);
    checkCudaErrors(hipFree(d_histBins));
}
