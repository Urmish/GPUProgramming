#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
#include <stdio.h>
int get_max_size (int a, int d)
{
    int temp = a/d;
    if (a%d != 0)
    {
        temp = temp+1;
    }
    return temp;
}
__global__ void createHistogramSlow ( unsigned int* d_bins,
                            unsigned int* const d_inputVals,
                            const size_t numElems,
                            int compareAndValue)
{
    int myId = blockDim.x*blockIdx.x + threadIdx.x;
    //int tid = threadIdx.x;
    if (myId < numElems)
    {
        if ((d_inputVals[myId] & compareAndValue) != 0)
        {
            atomicAdd(&d_bins[1], 1);
        }
        else
        {
            atomicAdd(&d_bins[0], 1);
        }
    }
}

__global__ void localHistograms (const unsigned int *input,
                                 unsigned int *output,
                                 const int size,
                                 int numBins,
                                 int perThreadReads,
                                 unsigned int compareAndValue,
                                 unsigned int *d_setOneIfOne)
{
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    for (int i=0;i<perThreadReads;i++)
    {
        if (myX*perThreadReads+i < size)
        {
            if ((input[myX*perThreadReads + i] & compareAndValue) != 0)
            {
                //Write to global Value
                output[myX*numBins + 1] = output[myX*numBins + 1]+1;
                d_setOneIfOne[myX] = 1;
            }
            else
            {
                //Write to global Value
                output[myX*numBins] = output[myX*numBins]+1;
                d_setOneIfOne[myX] = 0;
            }
        }
    }
}

__global__ void histogramReduce (int numBins,
                                 unsigned int *input,
                                 unsigned int *output,
                                 int size
                                )
{
    extern __shared__ unsigned int sdata3[];
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    for (int i=0;i<numBins;i++)
    {
        int index = myX*numBins+i;
        if (myX >= size)
        {
            sdata3[tid*numBins+i] = 0;
        }
        else
        {
            sdata3[tid*numBins+i] = input[index];
        }
    }
    __syncthreads();
    
    if (myX >= size)
    {
        if (tid == 0)
        {
            for (int i=0;i < numBins;i++)
            {
                output[blockIdx.x*numBins+i] = 0;
            }
        }
        return;
    }
    
    for (unsigned int s = blockDim.x/2; s > 0; s/=2)
    {
        if(tid < s)
        {
            for (int i=0;i<numBins;i++)
            {
                sdata3[tid*numBins+i] = sdata3[tid*numBins+i] + sdata3[tid*numBins+i+s*numBins];
            }
        }
        __syncthreads();
    }
    if (tid==0)
    {
        for (int i=0;i<numBins;i++)
        {
            //printf("Writing %d for bin value %d\n",sdata3[i],i);
            output[blockIdx.x*numBins+i] = sdata3[i];
        }
    }
}

void fastHistogram(unsigned int *d_finalHist, 
              unsigned int* const d_inputVals,
              const size_t numElems,
              unsigned int compareAndValue,
              unsigned int* d_setOneIfOne)
{
    int numBins = 2;
    unsigned int *d_histBins;
    int numBlocksPerGrid = 1024;
    int size = numElems;
    dim3 blockDim(1);
    dim3 gridDim(numBlocksPerGrid);
    int sizeOfBins = numBins*sizeof(unsigned int)*numBlocksPerGrid;
    int perThreadReads = get_max_size(size,numBlocksPerGrid);
    //Initialize temperory array variable
    checkCudaErrors(hipMalloc(&d_histBins, sizeOfBins));
    checkCudaErrors(hipMemset(d_histBins, 0, sizeOfBins));
    
    //Call first kernel
    localHistograms<<<gridDim,blockDim>>>(d_inputVals,d_histBins,size,numBins,perThreadReads,compareAndValue,d_setOneIfOne);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    
    size = numBlocksPerGrid;
    int localHistThreadsPerBlock = numBlocksPerGrid;
    dim3 blockDimLocalHist(localHistThreadsPerBlock);   
    unsigned int* d_curr_in;
    d_curr_in = d_histBins; 

    dim3 gridDimLocalHist(get_max_size(size,localHistThreadsPerBlock));
    histogramReduce<<<gridDimLocalHist,blockDimLocalHist,numBins*sizeof(unsigned int)*localHistThreadsPerBlock>>>(numBins,d_curr_in,d_finalHist,size);        
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());      
    checkCudaErrors(hipFree(d_curr_in));
}


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
     unsigned int* d_bins;
     unsigned int  h_bins[2];
     const size_t histo_size = 2*sizeof(unsigned int);
     checkCudaErrors(hipMalloc(&d_bins, histo_size));
     unsigned int* d_setOneIfOne;
     checkCudaErrors(hipMalloc(&d_setOneIfOne, numElems*sizeof(numElems)));
     for (int i=0;i<32;i++)
     {
         checkCudaErrors(hipMemset(d_bins, 0, histo_size));
         int compareAndValue = 1 << i;
         fastHistogram(d_bins,d_inputVals,numElems,compareAndValue,d_setOneIfOne);
         checkCudaErrors(hipMemcpy(&h_bins, d_bins, histo_size, hipMemcpyDeviceToHost));
         printf("Histogram Values - %d %d %d %d %d \n", h_bins[0], h_bins[1], h_bins[0]+h_bins[1], numElems, compareAndValue);
     }
     checkCudaErrors(hipFree(d_bins));
}

