#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
#include <stdio.h>
int get_max_size (int a, int d)
{
    int temp = a/d;
    if (a%d != 0)
    {
        temp = temp+1;
    }
    return temp;
}
__global__ void createHistogram ( unsigned int* d_bins,
                            unsigned int* const d_inputVals,
                            const size_t numElems,
                            int compareAndValue)
{
    int myId = blockDim.x*blockIdx.x + threadIdx.x;
    //int tid = threadIdx.x;
    if (myId < numElems)
    {
        if ((d_inputVals[myId] & compareAndValue) != 0)
        {
            atomicAdd(&d_bins[1], 1);
        }
        else
        {
            atomicAdd(&d_bins[0], 1);
        }
    }
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
     unsigned int* d_bins;
     unsigned int  h_bins[2];
     const size_t histo_size = 2*sizeof(unsigned int);
     checkCudaErrors(hipMalloc(&d_bins, histo_size));
     for (int i=0;i<32;i++)
     {
         checkCudaErrors(hipMemset(d_bins, 0, histo_size));
         int compareAddValue = 1 << i;
         int numThreadsPerBlock = 512;
         dim3 blockDim(numThreadsPerBlock);
         dim3 gridDim(get_max_size(numElems,numThreadsPerBlock));
         createHistogram <<<gridDim, blockDim>>> (d_bins,d_inputVals,numElems,compareAddValue);
         hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
         // copy the histogram data to host
         checkCudaErrors(hipMemcpy(&h_bins, d_bins, histo_size, hipMemcpyDeviceToHost));
         printf("Histogram Values - %d %d %d %d %d \n", h_bins[0], h_bins[1], h_bins[0]+h_bins[1], numElems, compareAddValue);
     }
}

