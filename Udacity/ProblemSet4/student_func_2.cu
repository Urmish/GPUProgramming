#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "reference_calc.cpp"
#include "utils.h"

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */
#include <stdio.h>

#define BLOCK_SIZE 512
int get_max_size (int a, int d)
{
    int temp = a/d;
    if (a%d != 0)
    {
        temp = temp+1;
    }
    return temp;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void fixup(unsigned int *input, unsigned int *aux, int len) {
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;
    if (blockIdx.x > 0) {
       if (start + t < len)
          input[start + t] += aux[blockIdx.x ];
       if (start + BLOCK_SIZE + t < len)
          input[start + BLOCK_SIZE + t] += aux[blockIdx.x ];
    }
}

__global__ void scanPart1 (unsigned int* input,
			   unsigned int* output,
			   unsigned int* aux,
			   int numElems)
{
	extern __shared__ unsigned int sdata[];
	//int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	int myLocalId = threadIdx.x;
	int start = 2 * blockIdx.x * BLOCK_SIZE; //Each block reads 2*BLOCK_SIZE so idx*this value is total inputs read
	int lastReadValue = 0;
	//Input Read
	if (start + myLocalId < numElems)
	{
       		sdata[myLocalId] = input[start + myLocalId];
	}
	else
	{
       		sdata[myLocalId] = 0;
	}
	if (start + BLOCK_SIZE + myLocalId  < numElems)
	{
       		sdata[BLOCK_SIZE + myLocalId] = input[start + BLOCK_SIZE + myLocalId];
	}
    	else
	{
       		sdata[BLOCK_SIZE + myLocalId] = 0;
	}
	__syncthreads();
	lastReadValue = sdata[2*BLOCK_SIZE-1];
	
	//Reduction
	int stride;
	for (stride = 1; stride <= BLOCK_SIZE; stride <<= 1) 
	{
       		int index = (myLocalId + 1) * stride * 2 - 1;
	        if (index < 2 * BLOCK_SIZE)
 	        	sdata[index] += sdata[index - stride];
	        __syncthreads();
	}
	if (myLocalId == 0)
	{
		sdata[2*BLOCK_SIZE-1] = 0;
	}
	__syncthreads();
	// Post reduction
	for (stride = BLOCK_SIZE ; stride; stride >>= 1) 
	{
       		int index = (myLocalId + 1) * stride * 2 - 1;
	        //if (index + stride < 2 * BLOCK_SIZE)
	        if (index < 2 * BLOCK_SIZE)
		{
//			unsigned int temp = sdata[index+stride];
//         		sdata[index + stride] += sdata[index];
//			sdata[index] = temp;
			unsigned int temp = sdata[index];
         		sdata[index] += sdata[index-stride];
			sdata[index-stride] = temp;
		}
	        __syncthreads();
    	}

	if (start + myLocalId < numElems)
	       	output[start + myLocalId] = sdata[myLocalId];
	if (start + BLOCK_SIZE + myLocalId < numElems)
        	output[start + BLOCK_SIZE + myLocalId] = sdata[BLOCK_SIZE + myLocalId];

	if (myLocalId == 0 && aux!=NULL)
       		aux[blockIdx.x] = sdata[2 * BLOCK_SIZE - 1] + lastReadValue;
}

__global__ void splitInput(int compareAndValue,
			   unsigned int* input,
			   unsigned int* output,
			   int maxElements)
{
	int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	if (myGlobalId >= maxElements)
	{
		return;
	}
	if(((input[myGlobalId] & compareAndValue)) > 0)
	{
		//printf("%d. %d & %d is 0\n",myGlobalId,input[myGlobalId],compareAndValue);
		output[myGlobalId] = 0;
	}
	else
	{
		//printf("%d. %d & %d is 1\n",myGlobalId,input[myGlobalId],compareAndValue);
		output[myGlobalId] = 1;
	}
	//printf("%d. %d\n",myGlobalId,input[myGlobalId]);
}

__global__ void possibleLocations (unsigned int* input,
				   unsigned int* input_scan,
				   unsigned int* output,
				   unsigned int numElems,
				   unsigned int compareAndValue)
{
	int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	int myLocalId = threadIdx.x;
	int start = 2 * blockIdx.x * BLOCK_SIZE; //Each block reads 2*BLOCK_SIZE so idx*this value is total inputs read
	int total = input_scan[numElems-1] + (((input[numElems-1] & compareAndValue) > 0)?0:1);
	//printf("Total %d\n",total);
	if (myLocalId + start < numElems)
	{
		//output[myGlobalId] = myGlobalId - input_scan[myGlobalId] + total;
		output[start + myLocalId] = start + myLocalId  - input_scan[start + myLocalId] + total;
		//printf("%d. %d might go to %d\n",start + myLocalId,input[myLocalId + start], output[start + myLocalId]);
	}
	if (myLocalId + start + BLOCK_SIZE < numElems)
	{
		output[start + myLocalId + BLOCK_SIZE] = start + myLocalId + BLOCK_SIZE - input_scan[start + myLocalId + BLOCK_SIZE] + total ;
		//printf("%d. %d might go to %d\n",start + myLocalId,input[myLocalId + start+BLOCK_SIZE], output[start + myLocalId + BLOCK_SIZE]);
	}
	
}


__global__ void finalLocations (   unsigned int* input,
				   unsigned int* input_scan,
				   unsigned int* input_vals,
				   unsigned int* d_setOneIfOne,
				   unsigned int* output,
				   unsigned int numElems,
                   unsigned int* inputPos,
                   unsigned int* outputPos)
{
	int myGlobalId = blockDim.x*blockIdx.x + threadIdx.x;
	int myLocalId = threadIdx.x;
	int start = 2 * blockIdx.x * BLOCK_SIZE; //Each block reads 2*BLOCK_SIZE so idx*this value is total inputs read
	if (myLocalId + start < numElems)
	{
		if (d_setOneIfOne[myLocalId + start] == 0)
		{
			output[input[myLocalId + start]] = input_vals[myLocalId + start];
            outputPos[input[myLocalId + start]] = inputPos[myLocalId + start];
			//printf("%d. %d goes to %d\n",myGlobalId, input_vals[myLocalId + start], input[myLocalId + start]);
		}	
		else
		{
			output[input_scan[myLocalId + start]] = input_vals[myLocalId + start];
            outputPos[input_scan[myLocalId + start]] = inputPos[myLocalId + start];
			//printf("%d. %d goes to %d\n",myGlobalId, input_vals[myLocalId + start], input_scan[myLocalId + start]);
		}
	}
	if (myLocalId + start + BLOCK_SIZE < numElems)
	{	
		if (d_setOneIfOne[myLocalId + start + BLOCK_SIZE] == 0)
		{
			output[input[myLocalId + start + BLOCK_SIZE]] = input_vals[myLocalId + start + BLOCK_SIZE];
            outputPos[input[myLocalId + start + BLOCK_SIZE]] = inputPos[myLocalId + start + BLOCK_SIZE];
			//printf("%d. %d goes to %d\n",myGlobalId, input_vals[myLocalId + start+ BLOCK_SIZE], input[myLocalId + start +BLOCK_SIZE]);
		}	
		else
		{
			output[input_scan[myLocalId + start + BLOCK_SIZE]] = input_vals[myLocalId + start + BLOCK_SIZE];
            outputPos[input_scan[myLocalId + start + BLOCK_SIZE]] = inputPos[myLocalId + start + BLOCK_SIZE];
			//printf("%d. %d goes to %d\n", myGlobalId, input_vals[myLocalId + start+BLOCK_SIZE] , input_scan[myLocalId + start +BLOCK_SIZE]);
		}
	}
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
    unsigned int* d_setOneIfOne;
    unsigned int* d_possibleLocations;
    //unsigned int* d_finalLocations;
    unsigned int* d_scan;
    unsigned int* h_scan;
    //unsigned int* d_inputVals;
	h_scan = (unsigned int*)malloc(numElems*sizeof(unsigned int));
	gpuErrchk(hipMalloc(&d_setOneIfOne, numElems*sizeof(unsigned int)));
    //gpuErrchk(hipMalloc(&d_inputVals, numElems*sizeof(unsigned int)));
	gpuErrchk(hipMalloc(&d_scan, numElems*sizeof(unsigned int)));
	gpuErrchk(hipMalloc(&d_possibleLocations, numElems*sizeof(unsigned int)));
    unsigned int* h_setOneIfOne;
    //d_inputVals = d_inputVals_2;
	h_setOneIfOne = (unsigned int*)malloc(numElems*sizeof(unsigned int));
	for (int i=0;i<32;i++)
    {
		 //gpuErrchk(hipMalloc(&d_finalLocations, numElems*sizeof(unsigned int)));
		 //printf("Round %d\n",i);
 	     gpuErrchk(hipMemset(d_setOneIfOne,0, numElems*sizeof(unsigned int)));
 	     gpuErrchk(hipMemset(d_scan,0, numElems*sizeof(unsigned int)));
	     int compareAndValue = 1 << i;
		 int numberThreadPerBlock = 512;
		 dim3 blockDim_si(numberThreadPerBlock);
		 dim3 gridDim_si(get_max_size(numElems,numberThreadPerBlock));
		 splitInput<<<gridDim_si,blockDim_si>>>(compareAndValue, d_inputVals, d_setOneIfOne, numElems);
	     //    gpuErrchk(hipMemcpy(h_setOneIfOne, d_setOneIfOne, numElems*sizeof(unsigned int), hipMemcpyDeviceToHost));
		 //for (int i=0;i<10;i++)
		 //{
		 //	printf("%d ", h_setOneIfOne[i]);
		 //	h_setOneIfOne[i] = 0;
		 //}
		 //printf("\n");
		 dim3 blockDim_sp(BLOCK_SIZE);
		 dim3 gridDim_sp(get_max_size(numElems,2*BLOCK_SIZE));
     	 unsigned int* d_aux;
     	 unsigned int* d_aux_scan;
		 unsigned int* h_aux;
	 	 gpuErrchk(hipMalloc(&d_aux, get_max_size(numElems,2*BLOCK_SIZE)*sizeof(unsigned int)));
	 	 gpuErrchk(hipMalloc(&d_aux_scan, get_max_size(numElems,2*BLOCK_SIZE)*sizeof(unsigned int)));
		 h_aux = (unsigned int*)malloc(get_max_size(numElems,2*BLOCK_SIZE)*sizeof(unsigned int));
		 //printf ("Size of Kernel is Grid - %d, Block - %d\n",gridDim_sp.x,blockDim_sp.x);
		 scanPart1<<<gridDim_sp,blockDim_sp,BLOCK_SIZE*2*sizeof(unsigned int)>>> (d_setOneIfOne,d_scan,d_aux,numElems);	
	     //    gpuErrchk(hipMemcpy(h_scan, d_scan, numElems*sizeof(numElems), hipMemcpyDeviceToHost));	
		 //for (int i=0;i<10;i++)
		 //{
		 //	printf("%d ", h_scan[i]);
		 //	h_scan[i] = 0;
		 //}
		 //printf("\n");
		 dim3 blockDim_sp2(get_max_size(numElems,2*BLOCK_SIZE));
	     //    gpuErrchk(hipMemcpy(h_aux, d_aux, blockDim_sp2.x*sizeof(unsigned int), hipMemcpyDeviceToHost));	
		 //for (int i=0;i<blockDim_sp2.x;i++)
		 //{
		 //	printf("%d ", h_aux[i]);
		 //	h_aux[i] = 0;
		 //}
		 //printf("\n");

		 //printf ("Size of Kernel is Grid - 1, Block - %d\n",blockDim_sp2.x);
		 scanPart1<<<1,blockDim_sp2,BLOCK_SIZE*2*sizeof(unsigned int)>>>(d_aux,d_aux_scan,NULL,blockDim_sp2.x);	
	     //gpuErrchk(hipMemcpy(h_aux, d_aux_scan, blockDim_sp2.x*sizeof(unsigned int), hipMemcpyDeviceToHost));	
		 //for (int i=0;i<blockDim_sp2.x;i++)
		 //{
		 //	printf("%d ", h_aux[i]);
		 //	h_aux[i] = 0;
		 //}
         
		 //printf("\n");
		 //printf ("Size of Kernel is Grid - %d, Block - %d\n",gridDim_sp.x,blockDim_sp.x);
		 fixup<<<gridDim_sp,blockDim_sp>>>(d_scan,d_aux_scan,numElems);
	     gpuErrchk(hipMemcpy(h_scan, d_scan, numElems*sizeof(unsigned int), hipMemcpyDeviceToHost));	
		 //for (int i=0;i<10;i++)
		 //{
		 //	printf("%d ", h_scan[i]);
		 //	h_scan[i] = 0;
		 //}
		 //printf("h_scan - %d\n",h_scan[numElems-1]);
		 //printf ("Size of Kernel is Grid - %d, Block - %d\n",gridDim_sp.x,blockDim_sp.x);
		 possibleLocations<<<gridDim_sp,blockDim_sp>>>(d_inputVals,d_scan, d_possibleLocations, numElems, compareAndValue);
         //        gpuErrchk(hipMemcpy(h_setOneIfOne, d_possibleLocations, numElems*sizeof(numElems), hipMemcpyDeviceToHost));
		 //printf ("Possible Locations are \n");
		 //for (int i=0;i<10;i++)
		 //{
		 //	printf("%d ", h_setOneIfOne[i]);
		 //	h_setOneIfOne[i] = 0;
		 //}
		 //printf ("\n");
		 //finalLocations<<<gridDim_sp,blockDim_sp>>>(d_possibleLocations,d_scan,d_inputVals, d_setOneIfOne, d_finalLocations,numElems,d_inputPos,d_outputPos);
         finalLocations<<<gridDim_sp,blockDim_sp>>>(d_possibleLocations,d_scan,d_inputVals, d_setOneIfOne, d_outputVals,numElems,d_inputPos,d_outputPos);
	 	 hipDeviceSynchronize(); 
		 //gpuErrchk(hipFree(d_inputVals));
		 //d_inputVals = d_finalLocations; 
         checkCudaErrors(hipMemcpy(d_inputPos, d_outputPos, numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));
         //checkCudaErrors(hipMemcpy(d_inputVals, d_finalLocations, numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));
         checkCudaErrors(hipMemcpy(d_inputVals, d_outputVals, numElems*sizeof(unsigned int), hipMemcpyDeviceToDevice));
         //gpuErrchk(hipMemcpy(h_setOneIfOne, d_finalLocations, numElems*sizeof(numElems), hipMemcpyDeviceToHost));
		 //printf ("\nFinal Positions are \n");
		 //for (int i=0;i<10;i++)
		 //{
		 //       printf("%d ", h_setOneIfOne[i]);
		 //       h_setOneIfOne[i] = 0;
		 //}

		 //printf("\n******************************************\n");
	         //printf("Histogram Values - %d %d %d %d %d \n", h_bins[0], h_bins[1], h_bins[0]+h_bins[1], numElems, compareAndValue);
    }
    //gpuErrchk(hipMemcpy(d_outputVals, d_finalLocations, numElems*sizeof(numElems), hipMemcpyDeviceToDevice));
    gpuErrchk(hipFree(d_setOneIfOne));
    free(h_setOneIfOne);
    //gpuErrchk(hipFree(d_possibleLocations));
    free(h_scan);
}
