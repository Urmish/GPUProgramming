#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include "stdio.h"

int get_max_size(int n, int d) {
    int size = n/d;
    if (n%d !=0 )
    {
        size = size+1;
    }
    return size;
}

__global__ void localHistograms (const unsigned int* const input,
                                 unsigned int *output,
                                 const int size,
                                 int numBins,
                                 int perThreadReads)
{
    extern __shared__ unsigned int sdata2[];
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    //int tid = threadIdx.x;
    //printf("myX is %d and myX*perThreadReads is %d and size is %d\n",myX,myX*perThreadReads,size);
    sdata2[threadIdx.x] = 0;

    if(myX < size)
    {
        atomicAdd(&sdata2[input[myX]],1);
    }
    
    output[blockIdx.x*numBins+threadIdx.x] = sdata2[threadIdx.x];
    
}


__global__
void histoReduce(int numBins,
                 unsigned int *input,
                 unsigned int *output,
                 int size,
                 int numHistReducePerBlock)
{
    extern __shared__ unsigned int sdata3[];
    int myX = blockDim.x*blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    for (int i=0;i<numHistReducePerBlock;i++)
    {
        int index = (blockIdx.x*numHistReducePerBlock+i)*numBins+threadIdx.x;
        if (blockIdx.x+i >= size)
        {
            sdata3[i*numBins + i] = 0;
        }
        else
        {
            sdata3[i*numBins + i] = input[index];
        }
    }
    __syncthreads();
    
    if (myX >= size)
    {
        output[blockIdx.x*numBins+threadIdx.x] = 0;
    }
    
    for (unsigned int s = numHistReducePerBlock/2; s > 0; s/=2)
    {
        for (int i=0;i<s;i++)
        {
            sdata3[i*numBins+tid] = sdata3[i*numBins+tid] + sdata3[i*numBins+s*numBins+tid];
        }
        __syncthreads();
    }
    
    output[blockIdx.x*numBins+threadIdx.x] = sdata3[threadIdx.x];
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
    unsigned int *d_histBins;
    int numThreadsPerBlock = 1024;
    printf("Size is %d\n",numElems);
    int perThreadReads = 1;
    dim3 blockDim(numThreadsPerBlock);
    dim3 gridDim(get_max_size(numElems,numThreadsPerBlock));
    int sizeOfBins = numBins*sizeof(unsigned int)*gridDim.x;
    printf("Grid Dimension %d\n",gridDim.x);
    checkCudaErrors(hipMalloc(&d_histBins, sizeOfBins));
    printf("Numbins - %d\n",numBins);
    localHistograms<<<gridDim,blockDim,numBins*sizeof(unsigned int)>>>(d_vals,d_histBins,numElems,numBins,perThreadReads);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    int size = gridDim.x;
    int localHistThreadsPerBlock = 1024;
    dim3 blockDimLocalHist(localHistThreadsPerBlock);
    
    unsigned int* d_curr_in;
    d_curr_in = d_histBins;
    unsigned int* d_curr_out;
    int numHistReducePerBlock = 8;
    while (size != 1 )
    {
        dim3 gridDimLocalHist(get_max_size(size,numHistReducePerBlock));
        int sharedMemorySize = numBins*sizeof(unsigned int)*numHistReducePerBlock;
        printf("Histogram Reduce - Block Size - %d with size - %d and shmem - %d\n",gridDimLocalHist.x,size,sharedMemorySize);
        //Allocate d_curr_out here
        checkCudaErrors(hipMalloc(&d_curr_out, numBins*sizeof(unsigned int) * gridDimLocalHist.x));
        
        //Call Kernel
        histoReduce<<<gridDimLocalHist,blockDimLocalHist,sharedMemorySize>>>(numBins,d_curr_in,d_curr_out,size,numHistReducePerBlock);    
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        
        checkCudaErrors(hipFree(d_curr_in));
        d_curr_in = d_curr_out;
        
        //Update size here
        size = get_max_size(size,numHistReducePerBlock);
    }
    //checkCudaErrors(hipFree(d_histBins));
    checkCudaErrors(hipMemcpy(d_histo, d_curr_out, sizeof(unsigned int)*numBins, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipFree(d_curr_out));
}
